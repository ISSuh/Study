#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

void verify1D(float * N, float  * P, float * mask, int width, int mask_width);

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

#define WIDTH 10000
#define MASK_WIDTH 5
#define O_TILE_WIDTH 1020
#define BLOCK_WIDTH (O_TILE_WIDTH + 4)

__global__ void Conv1D(float * N, float * P, float * Mask, int width, int mask_width)
{
	float output = 0.0f;
	int tx = threadIdx.x;
	int index_o = blockIdx.x*O_TILE_WIDTH + threadIdx.x;
	int index_i = index_o - 2;
	int index_m = 0;
	__shared__ float Ns[BLOCK_WIDTH];


	if (0 <= index_i && index_i <= width)
		Ns[tx] = N[index_i];
	else
		Ns[tx] = 0.0f;

	__syncthreads();

	if (tx < O_TILE_WIDTH) {
		for (int i = 0; i <  mask_width; i++) {
			output += Mask[i] * Ns[i+tx];
		}
		P[index_o] = output;
	}
}


int main()
{
	float *N, *P, *Mask;
	float *dev_N, *dev_P, *dev_Mask;

	N = (float*)malloc(sizeof(float)*WIDTH);
	P = (float*)malloc(sizeof(float)*WIDTH);
	Mask = (float*)malloc(sizeof(float) * 5);

	for (int i = 0; i < WIDTH; i++) {
		N[i] = (rand() % 100) / 100.00;
	}
	for (int i = 0; i<5; i++) {
		Mask[i] = (rand() % 100) / 100.00;
	}

	// Add vectors in parallel.
	HANDLE_ERROR(hipMalloc((void**)&dev_N, WIDTH * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_P, WIDTH * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_Mask, MASK_WIDTH * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(dev_N, N, WIDTH * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_Mask, Mask, MASK_WIDTH * sizeof(float), hipMemcpyHostToDevice));

	// Launch a kernel on the GPU with one thread for each element.
	dim3 dimBlock(BLOCK_WIDTH, 1, 1);
	dim3 dimGrid((WIDTH - 1) / O_TILE_WIDTH + 1, 1, 1);
	Conv1D << <dimGrid, dimBlock >> >(dev_N, dev_P, dev_Mask, WIDTH, MASK_WIDTH);
	HANDLE_ERROR(hipDeviceSynchronize());

	// Copy output vector from GPU buffer to host memory.
	HANDLE_ERROR(hipMemcpy(P, dev_P, WIDTH * sizeof(float), hipMemcpyDeviceToHost));

	verify1D(N, P, Mask, WIDTH, MASK_WIDTH);

	hipFree(dev_N);
	hipFree(dev_P);
	hipFree(dev_Mask);
	free(N);
	free(P);
	free(Mask);
	return 0;
}

void verify1D(float * N, float  * P, float * mask, int width, int mask_width)
{
	const float relativeTolerance = 1e-6;
	for (int i = 0; i < width; ++i) {
		float sum = 0.0f;
		for (int i_m = 0; i_m < mask_width; ++i_m) {
			int iN = i + i_m - (int)(mask_width / 2);
			if (iN >= 0 && iN < width) {
				sum += mask[i_m] * N[iN];
			}
		}
		float relativeError = (sum - P[i]) / sum;
		if (relativeError > relativeTolerance || relativeError < -relativeTolerance) {
			printf("TEST FAILED\n\n");
			exit(0);
		}
	}
	printf("TEST PASSED\n\n");
}