#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <time.h>
#include <math.h>
#include <string.h>

static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define WIDTH		256
#define HEIGHT		256
#define MR			3
#define MC			3
#define	MASK_N		3

const int N = 1024;
const int threadsPerBlock = 1024;
const int blocksperGrid = 5;

int MaskSobelX[3][3] = { { -1,0,1 },
{ -2,0,2 },
{ -1,0,1 } };

int MaskSobelY[3][3] = { { 1,2,1 },
{ 0,0,0 },
{ -1,-2,-1 } };
	
__global__ void Sobel_Conv(unsigned char *d_InImg, int *d_ConvX, int *d_ConvY, int width, int height, int mr, int mc, int size) {
	int outputX = 0, outputY = 0;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int i, j;

	int MaskSobelX[3][3] = { { -1,0,1 },
	{ -2,0,2 },
	{ -1,0,1 } };

	int MaskSobelY[3][3] = { { 1,2,1 },
	{ 0,0,0 },
	{ -1,-2,-1 } };

	if ((0 < row && row < height -1) && (0 < col && col < width-1)) {
		for (i = 0; i < mr; i++) {
			for (j = 0; j < mc; j++) {
				outputX += MaskSobelX[i][j] * d_InImg[(row + i - 1) * width + (col + j - 1)];
				outputY += MaskSobelY[i][j] * d_InImg[(row + i - 1) * width + (col + j - 1)];
			}
		}

		d_ConvX[row*width + col] = outputX;
		d_ConvY[row*width + col] = outputY;
	}

	else {
		d_ConvX[row*width + col] = 0;
		d_ConvY[row*width + col] = 0;
	}
}


__global__ void Detect_Edge(unsigned char *d_OrgImg, int *d_ConvX, int *d_ConvY,int *d_pImgSobel ,int width, int height,int *d_min,int *d_max) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int temp1, temp2;
	
	if ((row != 1 && row != height - 1) && (col != 1 && col != width - 1)) {
		if (d_ConvX[row*width + col] < 0)
			d_ConvX[row*width + col] = -d_ConvX[row*width + col];
		if (d_ConvY[row*width + col] < 0)
			d_ConvY[row*width + col] = -d_ConvY[row*width + col];

		d_pImgSobel[row*width + col] = d_ConvX[row*width + col] + d_ConvY[row*width + col];

		if (d_pImgSobel[row*width + col] < *d_min)
			*d_min = d_pImgSobel[row*width + col];
		if (d_pImgSobel[row*width + col] > *d_max)
			*d_max = d_pImgSobel[row*width + col];
	}

	__syncthreads();
						
	
	temp1 = (float)(255.0 / (*d_max - *d_min));
	temp2 = (float)(-255.0**d_min / (*d_max - *d_min));

	if ((row != 1 && row != height - 1) && (col != 1 && col != width - 1))
			d_OrgImg[row*width + col] = (unsigned char)(temp1*d_pImgSobel[row*width + col] + temp2);

}

int main()
{
	int min, max;
	int *dev_min, *dev_max;
	int *ConvX, *ConvY, *pImgSobel;
	int *dev_ConvX, *dev_ConvY,*dev_pImgSobel;
	float temp1, temp2;
	unsigned char *InImg;
	unsigned char *dev_InImg;
	unsigned char *OrgImg;
	unsigned char *dev_OrgImg;
	int newValue, Xval, Yval;
	int mr, mc;

	clock_t PL_start, PL_end;

	min = (int)10e10;
	max = (int)-10e10;

	InImg = (unsigned char*)malloc(sizeof(unsigned char)*(HEIGHT*WIDTH));
	OrgImg = (unsigned char*)malloc(sizeof(unsigned char)*(HEIGHT*WIDTH));

	FILE *infile = fopen("256x256.raw", "rb");
	fread(InImg, sizeof(unsigned char), HEIGHT * WIDTH, infile);
	fclose(infile);

	pImgSobel = (int*)malloc(sizeof(int)*(HEIGHT*WIDTH));
	ConvX = (int*)malloc(sizeof(int)*(HEIGHT*WIDTH));
	ConvY = (int*)malloc(sizeof(int)*(HEIGHT*WIDTH));

	memset(ConvX, 0, (HEIGHT*WIDTH) * sizeof(int));
	memset(ConvY, 0, (HEIGHT*WIDTH) * sizeof(int));
	memset(OrgImg, 0, (HEIGHT*WIDTH) * sizeof(unsigned char));
	memset(pImgSobel, 0, (HEIGHT*WIDTH) * sizeof(int));
	
	PL_start = clock();

	HANDLE_ERROR(hipMalloc((void**)&dev_InImg, WIDTH * HEIGHT * sizeof(unsigned char)));
	HANDLE_ERROR(hipMalloc((void**)&dev_ConvX, WIDTH * HEIGHT * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_ConvY, WIDTH * HEIGHT * sizeof(int)));

	HANDLE_ERROR(hipMemcpy(dev_InImg, InImg, WIDTH * HEIGHT * sizeof(unsigned char), hipMemcpyHostToDevice));
	
	Sobel_Conv << <blocksperGrid, threadsPerBlock >> > (dev_InImg, dev_ConvX, dev_ConvY, WIDTH, HEIGHT, MR, MC, WIDTH * HEIGHT);  
	HANDLE_ERROR(hipDeviceSynchronize());

	HANDLE_ERROR(hipMemcpy(ConvX, dev_ConvX, WIDTH * HEIGHT * sizeof(int), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(ConvY, dev_ConvY, WIDTH * HEIGHT * sizeof(int), hipMemcpyDeviceToHost));

	for (int i = 1000; i < HEIGHT; i++)
		for (int j = 1000; j < WIDTH; j++)
			printf("[%d][%d] %d %d\n",i,j,ConvX[i*WIDTH + j],ConvY[i*WIDTH + j]);

	/*for (int i = 1; i < HEIGHT - 1; i++) {
		for (int j = 1; j < WIDTH - 1; j++) {
			Xval = 0;
			Yval = 0;

			for (mr = 0; mr < MASK_N; mr++) {
				for (mc = 0; mc < MASK_N; mc++) {
					Xval += (MaskSobelX[mr][mc] * InImg[(i + mr - 1) * WIDTH + (j + mc - 1)]);
					Yval += (MaskSobelY[mr][mc] * InImg[(i + mr - 1) * WIDTH + (j + mc - 1)]);
				}
			}

			ConvX[i*WIDTH + j] = Xval;
			ConvY[i*WIDTH + j] = Yval;
		}
	}*/

	for (int i = 1; i < HEIGHT - 1; i++) {
		for (int j = 1; j < WIDTH - 1; j++) {

			temp1 = ConvX[i*WIDTH + j];
			temp2 = ConvY[i*WIDTH + j];

			if (temp1 < 0)
				temp1 = -temp1;
			if (temp2 < 0)
				temp2 = -temp2;

			pImgSobel[i*WIDTH + j] = temp1 + temp2;
		}
	}

	//�ִ밪,�ּҰ� Ž��
	for (int i = 1; i < HEIGHT - 1; i++) {
		for (int j = 1; j < WIDTH - 1; j++) {

			if (pImgSobel[i*WIDTH + j]<min)
				min = pImgSobel[i*WIDTH + j];
			if (pImgSobel[i*WIDTH + j]>max)
				max = pImgSobel[i*WIDTH + j];

		}
	}

	//��ȯ�� ������� �̸� ��� 
	temp1 = (float)(255.0 / (max - min));
	temp2 = (float)(-255.0*min / (max - min));

	for (int i = 1; i<HEIGHT - 1; i++) {
		for (int j = 1; j<WIDTH - 1; j++) {
			//[min,max]������ ���� [0,255]������ ��ȯ 
			newValue = pImgSobel[i*WIDTH + j];
			newValue = temp1 * newValue + temp2;
			OrgImg[i*WIDTH + j] = (unsigned char)newValue;
		}
	}

	
	/*HANDLE_ERROR(hipMalloc((void**)&dev_OrgImg, WIDTH * HEIGHT * sizeof(unsigned char)));
	HANDLE_ERROR(hipMalloc((void**)&dev_ConvX, WIDTH * HEIGHT * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_ConvY, WIDTH * HEIGHT * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_pImgSobel, WIDTH * HEIGHT * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_min, HEIGHT * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_max, HEIGHT * sizeof(int)));

	HANDLE_ERROR(hipMemset(dev_min,0,sizeof(int)));
	HANDLE_ERROR(hipMemset(dev_max, 0, sizeof(int)));

	HANDLE_ERROR(hipMemcpy(dev_ConvX, ConvX, WIDTH * HEIGHT * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_ConvY, ConvY, WIDTH * HEIGHT * sizeof(int), hipMemcpyHostToDevice));

	Detect_Edge << <HEIGHT, WIDTH>> > (dev_OrgImg, dev_ConvX, dev_ConvY, dev_pImgSobel, WIDTH, HEIGHT,dev_min,dev_max);

	HANDLE_ERROR(hipMemcpy(OrgImg, dev_OrgImg, WIDTH * HEIGHT * sizeof(unsigned char), hipMemcpyDeviceToHost));*/

	PL_end = clock();

	FILE *outfile = fopen("CS.raw", "wb");
	fwrite(OrgImg, sizeof(unsigned char), HEIGHT * WIDTH, outfile);
	fclose(outfile);

	printf("Finish\n");
	printf("Block per Grid			:  %d\n",blocksperGrid);
	printf("Thread per Block		:  %d\n",threadsPerBlock);
	printf("Average Parallel Runtime 	: %f\n", (float)(PL_end - PL_start) / CLOCKS_PER_SEC);

	hipFree(dev_ConvX);
	hipFree(dev_ConvY);
	//hipFree(dev_OrgImg);
	//hipFree(dev_pImgSobel);
	//hipFree(dev_min);
	//hipFree(dev_max);

	free(InImg);
	free(OrgImg);
	free(pImgSobel);
	free(ConvX);
	free(ConvY);
}