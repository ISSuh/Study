#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <time.h>
#include <math.h>

const int N = 1024;
const int threadsPerBlock = 1024;
const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

void verify2D(float * N, float  * P, float * mask, int width, int height, int mask_width);

#define WIDTH 300
#define HEIGHT 200
#define MASK_WIDTH 5

#define O_TILE_WIDTH 12
#define BLOCK_WIDTH (O_TILE_WIDTH + 4)

__global__ void Conv2D(float * N, float * P, float * Mask, int width, int height, int mask_width)
{
	float output = 0.0f;
	int tx = threadIdx.x; int ty = threadIdx.y;
	int row_o = blockIdx.y*O_TILE_WIDTH + threadIdx.y;
	int col_o = blockIdx.x*O_TILE_WIDTH + threadIdx.x;
	int row_i = row_o - 2;
	int col_i = col_o - 2;
	int i, j;
	int mask_i = 0, mask_j = 0;
	__shared__ float Ns[BLOCK_WIDTH][BLOCK_WIDTH];

	// All threads participate in loading data to Ns[][]
	if ((0 < row_i && row_i < height) && (0 < col_i && col_i < width))
		Ns[ty][tx] = N[width*row_i + col_i];
	else
		Ns[ty][tx] = 0.0f;

	__syncthreads();

	// Some threads participate in calculating P[]. Be careful around the boundaries of Matrix!
	if ((row_o < height && col_o < width) && (ty < O_TILE_WIDTH && tx < O_TILE_WIDTH)) {
		for (i = 0; i < mask_width ; i++) {
			mask_j = 0;
			for (j = 0; j < mask_width ; j++) {
				output += Mask[i*mask_width + j] * Ns[i+ty][j+tx];
			}
		}
		P[row_o*WIDTH + col_o] = output;
	}
}


int main()
{
	float *N, *P, *Mask;
	float *dev_N, *dev_P, *dev_Mask;

	N = (float*)malloc(sizeof(float)*WIDTH*HEIGHT);
	P = (float*)malloc(sizeof(float)*WIDTH*HEIGHT);
	Mask = (float*)malloc(sizeof(float)*MASK_WIDTH*MASK_WIDTH);

	for (int i = 0; i < WIDTH*HEIGHT; i++) { N[i] = (rand() % 100) / 100.00; }
	for (int i = 0; i<MASK_WIDTH*MASK_WIDTH; i++) { Mask[i] = (rand() % 100) / 100.00; }

	// Add vectors in parallel.
	HANDLE_ERROR(hipMalloc((void**)&dev_N, WIDTH * HEIGHT * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_P, WIDTH * HEIGHT * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_Mask, MASK_WIDTH * MASK_WIDTH * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(dev_N, N, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_Mask, Mask, MASK_WIDTH * MASK_WIDTH * sizeof(float), hipMemcpyHostToDevice));

	// Launch a kernel on the GPU with one thread for each element.
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 dimGrid((WIDTH - 1) / O_TILE_WIDTH + 1, (HEIGHT - 1) / O_TILE_WIDTH + 1, 1);
	Conv2D << <dimGrid, dimBlock >> >(dev_N, dev_P, dev_Mask, WIDTH, HEIGHT, MASK_WIDTH);
	HANDLE_ERROR(hipDeviceSynchronize());

	// Copy output vector from GPU buffer to host memory.
	HANDLE_ERROR(hipMemcpy(P, dev_P, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost));
	verify2D(N, P, Mask, WIDTH, HEIGHT, MASK_WIDTH);

	hipFree(dev_N);     hipFree(dev_P); hipFree(dev_Mask);
	free(N); free(P); free(Mask);
	return 0;
}

void verify2D(float * N, float  * P, float * mask, int width, int height, int mask_width) {
	const float relativeTolerance = 1e-6;
	for (int row = 0; row < height; ++row) {
		for (int col = 0; col < width; ++col) {
			float sum = 0.0f;
			for (int row_m = 0; row_m < mask_width; ++row_m) {
				for (int col_m = 0; col_m < mask_width; ++col_m) {
					int row_N = row + row_m - (int)(mask_width / 2);
					int col_N = col + col_m - (int)(mask_width / 2);
					if (col_N >= 0 && col_N < width && row_N >= 0 && row_N<height) {
						sum += mask[row_m*mask_width + col_m] * N[row_N*width + col_N];
					}
				}
			}
			float relativeError = (sum - P[row*width + col]) / sum;
			if (relativeError > relativeTolerance || relativeError < -relativeTolerance) {
				printf("TEST FAILED\n");
				exit(0);
			}
		}
	}
	printf("TEST PASSED\n\n");
}
