#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <Math.h>

static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

const int N = 1024;
const int threadsPerBlock = 1024;
const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

/*__global__ void scan_work_inefficient(long * dev_a, long * dev_output)
{
	__shared__ long SMem[threadsPerBlock];
	int tx = threadIdx.x;
	int stride;
	int temp;
	SMem[tx] = dev_a[tx];

	for (stride = 1; stride <= tx; stride *= 2) {

		__syncthreads();
		 temp = SMem[tx - stride];
		__syncthreads();

		SMem[tx] += temp;
	}
	
	dev_output[tx] = SMem[tx];
}

__global__ void scan_work_efficient(long * dev_a, long * dev_output )
{
	__shared__ long SMem[threadsPerBlock];
	int tx = threadIdx.x;
	int stride,index;

	SMem[2*tx] = dev_a[2*tx];
	SMem[2*tx+1] = dev_a[2*tx+1];

	// Reduction
	for(stride = 1 ; stride <= threadsPerBlock/2 ; stride *= 2 ){
		index = (threadIdx.x+1)*stride*2-1;
		if( index < threadsPerBlock )
			SMem[index] += SMem[index-stride];
		__syncthreads();
	}

	// Post reduction
	for(stride = threadsPerBlock/4 ; stride >0 ; stride /= 2 ){
		index = (threadIdx.x+1)*stride*2-1;
		if( index+stride < threadsPerBlock )
			SMem[index+stride] += SMem[index];
		__syncthreads();
	}

	dev_output[2*tx] = SMem[2*tx];
	dev_output[2*tx+1] = SMem[2*tx+1];
}*/

__global__ void scan_work_inefficient(long * dev_a, long * dev_output)
{
	__shared__ long SMem[threadsPerBlock];
	int tx = threadIdx.x;
	int stride;
	long temp = 0;
	SMem[tx] = dev_a[tx];

	for (stride = 1; stride <= tx; stride <<= 1) {

		__syncthreads();	
		if (tx >= stride)
			temp = SMem[tx - stride];
		__syncthreads(); 

		SMem[tx] += temp;
	}

	dev_output[tx] = SMem[tx];
}

__global__ void scan_work_efficient(long * dev_a, long * dev_output)
{
	__shared__ long SMem[threadsPerBlock];
	int tx = threadIdx.x;
	int index,stride;
	SMem[2 * tx] = dev_a[2 * tx];
	SMem[2 * tx + 1] = dev_a[2 * tx + 1];

	// Reduction
	for (stride = 1; stride < threadsPerBlock; stride <<= 1) {
		index = (threadIdx.x + 1)*stride * 2 - 1;
		if (index < threadsPerBlock)
			SMem[index] += SMem[index - stride];

		__syncthreads();
	}
	// Post reduction
	for (stride = (threadsPerBlock >> 2); stride > 0; stride >>= 1) {
		index = (threadIdx.x + 1)*stride * 2 - 1;
		if (index + stride < threadsPerBlock)
			SMem[index + stride] += SMem[index];

		__syncthreads();
	}

	dev_output[2 * tx] = SMem[2 * tx];
	dev_output[2 * tx + 1] = SMem[2 * tx + 1];
}

int main()
{
    long   *a, *output;
    long   *dev_a, *dev_output;

    // allocate memory on the cpu side
    a = (long*)malloc( N*sizeof(long) );
    output = (long*)malloc( N*sizeof(long) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N*sizeof(long) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_output, N*sizeof(long) ) );

    // fill in the host memory with data
    for (int i=0; i<N; i++) {
        a[i] = (rand()%10);
    }
    printf("blocksPerGrid:%d , threadsPerBLock:%d\n", blocksPerGrid, threadsPerBlock );

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N*sizeof(long), hipMemcpyHostToDevice ) );
  
    clock_t start = clock();
	//scan_work_inefficient <<<blocksPerGrid,threadsPerBlock>>>( dev_a, dev_output );
	scan_work_efficient << <blocksPerGrid, threadsPerBlock / 2 >> >(dev_a, dev_output);

    HANDLE_ERROR( hipDeviceSynchronize());
    clock_t end = clock();

    printf(" kernel execution time: %lf sec\n", (double)(end-start)/(double)CLOCKS_PER_SEC );

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( output, dev_output, N*sizeof(long), hipMemcpyDeviceToHost ) );

	long d = 0;
	for (int i = 0; i<N; i++) {
		d += a[i];
		if (d != output[i]) {
			printf("i=%d, d:%d and output:%d\n", i, d, output[i]);
			printf("Error!"); goto CLEANUP;
		}
	}
	printf("PASS!\n");

CLEANUP:
	// free memory on the gpu side
	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_output));

	// free memory on the cpu side
	free(a);
	free(output);
}


