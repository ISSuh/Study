#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#define LEN_M 4
#define LEN_N 4
#define LEN_K 3

#define TILE_WIDTH 2

__global__ void MatrixMulOnDevice(int m, int n, int k, float* A, float* B, float* C)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	if ((Row < m) && (Col < k))
	{
		float Cvalue = 0.0;
		for (int i = 0; i < n; ++i)
			Cvalue += A[Row*n + i] * B[Col + i*k];
		C[Row*k + Col] = Cvalue;
	}
}


int main()
{
	// Allocate and initialize the matrices A, B, C
	float * A, *B, *C;

	A = (float*)malloc(LEN_M*LEN_N * sizeof(float));
	B = (float*)malloc(LEN_N*LEN_K * sizeof(float));
	C = (float*)malloc(LEN_M*LEN_K * sizeof(float));

	for (int i = 0; i<LEN_M*LEN_N; i++) A[i] = i;
	for (int i = 0; i<LEN_N*LEN_K; i++) B[i] = i;
	for (int i = 0; i<LEN_M*LEN_K; i++) C[i] = 0.0;

	// I/O to read the input matrices A and B
	float * dev_A, *dev_B, *dev_C;
	hipMalloc((void**)&dev_A, LEN_M*LEN_N * sizeof(float));
	hipMalloc((void**)&dev_B, LEN_N*LEN_K * sizeof(float));
	hipMalloc((void**)&dev_C, LEN_M*LEN_K * sizeof(float));

	hipMemcpy(dev_A, A, LEN_M*LEN_N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, LEN_N*LEN_K * sizeof(float), hipMemcpyHostToDevice);

	// A*B on the device
	dim3 dimGrid((LEN_K - 1) / TILE_WIDTH + 1, (LEN_M - 1) / TILE_WIDTH + 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
	MatrixMulOnDevice << <dimGrid, dimBlock >> >(LEN_M, LEN_N, LEN_K, dev_A, dev_B, dev_C);

	// I/O to write the output matrix C
	hipMemcpy(C, dev_C, LEN_M*LEN_K * sizeof(float), hipMemcpyDeviceToHost);

	printf("C:");
	for (int i = 0; i<LEN_M*LEN_K; i++) printf(" %6.1f", C[i]);
	printf("\n");

	// Free matrices A, B, C
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);

	free(A);
	free(B);
	free(C);
	return 0;
}

