#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <time.h>

#define LEN_M (2*1024+3)
#define LEN_N (2*1024+3)
#define LEN_K (1*1024+3)
#define TILE_WIDTH 32

__global__ void MatrixMulOnDevice(int m, int n, int k, float * A, float * B, float * C) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	if (col < LEN_K && row < LEN_M)
		for (int i = 0; i < n; ++i)
			C[row * k + col] += A[row * n + i] * B[col + i*k];
}

__global__ void TiledMatrixMulKernel(int m, int n, int k, float* A, float* B, float* C)
{
	__shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	int Row = by * blockDim.y + ty;
	int Col = bx * blockDim.x + tx;

	float Cvalue = 0;
	for (int t = 0; t < n / TILE_WIDTH; ++t) {	// iterate over phases
												// load A and B tiles into shared memory 
		if ((Row < m) && (t*TILE_WIDTH + tx < n))
			ds_A[ty][tx] = A[Row*n + (t*TILE_WIDTH + tx)];
		else
			ds_A[ty][tx] = 0;

		if ((t*TILE_WIDTH + ty < n) && (Col < k))
			ds_B[ty][tx] = B[(t*TILE_WIDTH + ty)*k + Col];
		else
			ds_B[ty][tx] = 0;

		__syncthreads();												// ���ſ� �Լ�

																		// compute Cvalue based on ds_A[][] and ds_B[][]

		for (int i = 0; i < TILE_WIDTH; i++)
			Cvalue += ds_A[ty][i] * ds_B[i][tx];

		__syncthreads();
	}

	if (Row < m && Col < k)
		C[Row*k + Col] = Cvalue;
}

void MatrixMulOnHost(int m, int n, int k, float* A, float* B, float* C)
{
	for (int Row = 0; Row < m; ++Row)
		for (int Col = 0; Col < k; ++Col) {
			float sum = 0;

			for (int i = 0; i < n; ++i) {
				float a = A[Row * n + i];
				float b = B[Col + i*k];
				sum += a * b;
			}

			C[Row * k + Col] = sum;
		}
}

int main()
{
	clock_t start, stop, start1, stop1;
	// Allocate and initialize the matrices A, B, C
	float * A, *B, *C, *D;

	A = (float*)malloc(LEN_M*LEN_N * sizeof(float));
	B = (float*)malloc(LEN_N*LEN_K * sizeof(float));
	C = (float*)malloc(LEN_M*LEN_K * sizeof(float));
	D = (float*)malloc(LEN_M*LEN_K * sizeof(float));

	for (int i = 0; i<LEN_M*LEN_N; i++) A[i] = i % 3;
	for (int i = 0; i<LEN_N*LEN_K; i++) B[i] = i % 4;
	for (int i = 0; i<LEN_M*LEN_K; i++) C[i] = 0.0;
	for (int i = 0; i<LEN_M*LEN_K; i++) D[i] = 0.0;

	// I/O to read the input matrices A and B
	float * dev_A, *dev_B, *dev_C;
	hipMalloc((void**)&dev_A, LEN_M*LEN_N * sizeof(float));
	hipMalloc((void**)&dev_B, LEN_N*LEN_K * sizeof(float));
	hipMalloc((void**)&dev_C, LEN_M*LEN_K * sizeof(float));

	hipMemcpy(dev_A, A, LEN_M*LEN_N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, LEN_N*LEN_K * sizeof(float), hipMemcpyHostToDevice);

	// A*B on the device
	dim3 dimGrid((LEN_K - 1) / TILE_WIDTH + 1, (LEN_M - 1) / TILE_WIDTH + 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

	start = clock();
	MatrixMulOnDevice << <dimGrid, dimBlock >> >(LEN_M, LEN_N, LEN_K, dev_A, dev_B, dev_C);
	hipDeviceSynchronize();
	stop = clock();

	printf("Run time(no tile) : : %lf sec\n", (float)(stop - start) / (float)CLOCKS_PER_SEC);


	start1 = clock();
	TiledMatrixMulKernel << <dimGrid, dimBlock >> >(LEN_M, LEN_N, LEN_K, dev_A, dev_B, dev_C);
	hipDeviceSynchronize();
	stop1 = clock();

	printf("Run time(tile) : : %lf sec\n", (float)(stop1 - start1) / (float)CLOCKS_PER_SEC);

	// I/O to write the output matrix C
	hipMemcpy(C, dev_C, LEN_M*LEN_K * sizeof(float), hipMemcpyDeviceToHost);

	printf("C:");
	for (int i = 0; i<LEN_M*LEN_K; i++) printf(" %6.1f", C[i]);
	printf("\n");
	printf("\n");

	MatrixMulOnHost(LEN_M, LEN_N, LEN_K, A, B, D);

	printf("D:");
	for (int i = 0; i<LEN_M*LEN_K; i++) printf(" %6.1f", D[i]);
	printf("\n");

	// Free matrices A, B, C
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);

	free(A);
	free(B);
	free(C);
	return 0;
}