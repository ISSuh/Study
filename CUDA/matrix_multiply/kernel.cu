#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

#define LEN_M 4
#define LEN_N 4
#define LEN_K 3

#define TILE_WIDTH 2

__global__ void MatrixMulKernel(int m, int n, int k, float* A, float* B, float* C)
{
	int Row = threadIdx.y + blockDim.y * blockIdx.y;        // row index of C matrix
	int Col = threadIdx.x + blockDim.x * blockIdx.x;        // col index of C matrix
	int i;
	float val = 0.0;

	if (Row < m && Col < k)  // check boundary
	{
		for (i = 0; i < n; i++) {
			C[k*Row + Col] += A[n*Row + i] * B[i*k + Col];
		}
	}
}


int main()
{
	// Allocate and initialize the matrices A, B, C
	float * A, *B, *C;

	A = (float*)malloc(LEN_M*LEN_N * sizeof(float));
	B = (float*)malloc(LEN_N*LEN_K * sizeof(float));
	C = (float*)malloc(LEN_M*LEN_K * sizeof(float));

	for (int i = 0; i<LEN_M*LEN_N; i++) A[i] = i;
	for (int i = 0; i<LEN_N*LEN_K; i++) B[i] = i;
	for (int i = 0; i<LEN_M*LEN_K; i++) C[i] = 0.0;

	// I/O to read the input matrices A and B
	float * dev_A, *dev_B, *dev_C;
	hipMalloc((void**)&dev_A, LEN_M*LEN_N * sizeof(float));
	hipMalloc((void**)&dev_B, LEN_N*LEN_K * sizeof(float));
	hipMalloc((void**)&dev_C, LEN_M*LEN_K * sizeof(float));


	hipMemcpy(dev_A, A, LEN_M*LEN_N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, LEN_N*LEN_K * sizeof(float), hipMemcpyHostToDevice);

	// A*B on the device
	dim3 dimGrid((LEN_K - 1) / TILE_WIDTH + 1, (LEN_M - 1) / TILE_WIDTH + 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
	MatrixMulKernel << <dimGrid, dimBlock >> >(LEN_M, LEN_N, LEN_K, dev_A, dev_B, dev_C);

	// I/O to write the output matrix C
	hipMemcpy(C, dev_C, LEN_M*LEN_K * sizeof(float), hipMemcpyDeviceToHost);

	printf("C:");
	for (int i = 0; i<LEN_M*LEN_K; i++) printf(" %6.1f", C[i]);
	printf("\n");

	// Free matrices A, B, C
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);

	free(A);
	free(B);
	free(C);
	return 0;
}

